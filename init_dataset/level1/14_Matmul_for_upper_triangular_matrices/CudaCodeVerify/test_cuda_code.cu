#include "hip/hip_runtime.h"
// module_fn.cu

#include <torch/extension.h>

__global__ void matmul_upper_triangular_kernel(const float* A, const float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N && col >= row) {
        float sum = 0.0f;
        for (int k = row; k < N; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

torch::Tensor matmul_upper_triangular(torch::Tensor A, torch::Tensor B) {
    int N = A.size(0);
    torch::Tensor C = torch::zeros_like(A);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + 15) / 16, (N + 15) / 16);

    matmul_upper_triangular_kernel<<<numBlocks, threadsPerBlock>>>(A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>(), N);

    hipDeviceSynchronize();
    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &matmul_upper_triangular, "Upper triangular matrix multiplication");
}