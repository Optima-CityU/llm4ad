#include "hip/hip_runtime.h"
// Matrix multiplication CUDA kernel
#include <torch/extension.h>

__global__ void matmul_kernel(const float* A, const float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float value = 0.0f;
        for (int k = 0; k < N; ++k) {
            value += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = value;
    }
}

torch::Tensor matmul_cuda(torch::Tensor A, torch::Tensor B) {
    int N = A.size(0);
    auto C = torch::zeros({N, N}, A.options());

    const dim3 block(16, 16);
    const dim3 grid((N + block.x - 1) / block.x, (N + block.y - 1) / block.y);

    matmul_kernel<<<grid, block>>>(A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>(), N);
    hipDeviceSynchronize();

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &matmul_cuda, "Matrix multiplication kernel");
}